
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(unsigned char *p)
{
	int i = threadIdx.x;
	int j = threadIdx.y;
	p[i * 9 + j] = (i + 1) * (j + 1);
}

int main( void )
{
	unsigned char table[81];
	unsigned char *mem;
	hipMalloc((void **)&mem, 81);
	dim3 t(9, 9);
	kernel<<<1,t>>>(mem);
	hipMemcpy(table, mem, 81, hipMemcpyDeviceToHost);
	hipFree(mem);

	for (int i = 0; i < 9; i++) {
		for (int j = 0; j < 9; j++) {
			printf(" %2d", table[i * 9 + j]);
		}
		putchar('\n');
	}

	return 0;
}
